
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

#define N 80000
#define MAX_ERR 1e-6

//  __global__ defines a kernel function
__global__ void vector_add(float *out, float *a, float *b, int n) {
    for (int i=0; i < n; i++) out[i] = a[i] + b[i];
} 

int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    // Allocate memory on
    a = (float*)malloc(sizeof(float) * N);
    b = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for (int i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Allocate memory on device
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);


    // Transfer from host to device
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // vector_add(out, a, b, N);
    // use one block, one thread --> going parallel in the next exercise 
    vector_add<<<1,1>>>(d_out, d_a, d_b, N);
    // <<<# blocks, #threads in each block>>>


    // Transfer result from device to host
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Verification  
    for(int i = 0; i < N; i++){
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }
    printf("out[0] = %f\n", out[0]);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    free(a);
    free(b);
    free(out);


    return 0;
}